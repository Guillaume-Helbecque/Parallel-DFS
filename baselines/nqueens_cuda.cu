/*
  C+Cuda backtracking algorithm to solve instances of the N-Queens problem.
*/

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <time.h>
#include "hip/hip_runtime.h"

#define BLOCK_SIZE 512

#define MIN(a, b) ((a) < (b) ? (a) : (b))

/*******************************************************************************
Implementation of N-Queens Nodes.
*******************************************************************************/

#define MAX_QUEENS 21

typedef struct
{
  uint8_t depth;
  uint8_t board[MAX_QUEENS];
} Node;

void initRoot(Node* root, const int N)
{
  root->depth = 0;
  for (uint8_t i = 0; i < N; i++) {
    root->board[i] = i;
  }
}

/*******************************************************************************
Implementation of a dynamic-sized single pool data structure.
Its initial capacity is 1024, and we reallocate a new container with double
the capacity when it is full. Since we perform only DFS, it only supports
'pushBack' and 'popBack' operations.
*******************************************************************************/

#define CAPACITY 1024

typedef struct
{
  Node* elements;
  int capacity;
  int size;
} SinglePool;

void initSinglePool(SinglePool* pool)
{
  pool->elements = (Node*)malloc(CAPACITY * sizeof(Node));
  pool->capacity = CAPACITY;
  pool->size = 0;
}

void pushBack(SinglePool* pool, Node node)
{
  if (pool->size >= pool->capacity) {
    pool->capacity *= 2;
    pool->elements = (Node*)realloc(pool->elements, pool->capacity * sizeof(Node));
  }

  pool->elements[pool->size++] = node;
}

Node popBack(SinglePool* pool, int* hasWork)
{
  if (pool->size > 0) {
    *hasWork = 1;
    return pool->elements[--pool->size];
  }

  return (Node){0};
}

void deleteSinglePool(SinglePool* pool)
{
  free(pool->elements);
}

/*******************************************************************************
Implementation of the single-core single-GPU N-Queens search.
*******************************************************************************/

void parse_parameters(int argc, char* argv[], int* N, int* G, int* minSize, int* maxSize)
{
  if (argc != 5) {
    printf("Usage: %s <N> <g> <minSize> <maxSize>\n", argv[0]);
    exit(0);
  }

  *N = atoi(argv[1]);
  *G = atoi(argv[2]);
  *minSize = atoi(argv[3]);
  *maxSize = atoi(argv[4]);

  if ((*N <= 0) || (*G <= 0) || (*minSize <= 0) || (*maxSize <= 0)) {
    printf("All parameters must be positive integers.\n");
    exit(0);
  }
}

void print_settings(const int N, const int G)
{
  printf("\n=================================================\n");
  printf("Resolution of the %d-Queens instance using C+CUDA\n", N);
  printf("  with %d safety check(s) per evaluation\n", G);
  printf("=================================================\n");
}

void print_results(const unsigned long long int exploredTree,
  const unsigned long long int exploredSol, const double timer)
{
  printf("\n=================================================\n");
  printf("Size of the explored tree: %llu\n", exploredTree);
  printf("Number of explored solutions: %llu\n", exploredSol);
  printf("Elapsed time: %.4f [s]\n", timer);
  printf("=================================================\n");
}

void swap(uint8_t* a, uint8_t* b)
{
  uint8_t tmp = *b;
  *b = *a;
  *a = tmp;
}

// Check queen's safety.
int isSafe(const int G, const uint8_t* board, const uint8_t queen_num, const uint8_t row_pos)
{
  for (int g = 0; g < G; g++) {
    for (int i = 0; i < queen_num; i++) {
      const uint8_t other_row_pos = board[i];

      if (other_row_pos == row_pos - (queen_num - i) ||
          other_row_pos == row_pos + (queen_num - i)) {
        return 0;
      }
    }
  }

  return 1;
}

// Evaluate and generate children nodes on CPU.
void decompose(const int N, const int G, const Node parent,
  unsigned long long int* tree_loc, unsigned long long int* num_sol, SinglePool* pool)
{
  const uint8_t depth = parent.depth;

  if (depth == N) {
    *num_sol += 1;
  }
  for (int j = depth; j < N; j++) {
    if (isSafe(G, parent.board, depth, parent.board[j])) {
      Node child;
      memcpy(child.board, parent.board, N * sizeof(uint8_t));
      swap(&child.board[depth], &child.board[j]);
      child.depth = depth + 1;
      pushBack(pool, child);
      *tree_loc += 1;
    }
  }
}

// Evaluate a bulk of parent nodes on GPU.
__global__ void evaluate_gpu(const int N, const int G, const Node* parents_d, uint8_t* evals_d, const int size)
{
  int threadId = blockIdx.x * blockDim.x + threadIdx.x;

  if (threadId < size) {
    const int parentId = threadId / N;
    const int k = threadId % N;
    const Node parent = parents_d[parentId];
    const uint8_t depth = parent.depth;

    evals_d[threadId] = 1;

    // If child 'k' is not scheduled, we evaluate its safety 'G' times, otherwise 0.
    const int G_notScheduled = G * (k >= depth);
    for (int g = 0; g < G_notScheduled; g++) {
      for (int i = 0; i < depth; i++) {
        const uint8_t other_row_pos = parent.board[i];
        const int isNotSafe = (other_row_pos == parent.board[k] - (depth - i) ||
                               other_row_pos == parent.board[k] + (depth - i));

        evals_d[threadId] *= (1 - isNotSafe);
      }
    }
  }
}

// Generate children nodes (evaluated by GPU) on CPU.
void generate_children(const int N, const Node* parents, const int size, const uint8_t* evals,
  unsigned long long int* exploredTree, unsigned long long int* exploredSol, SinglePool* pool)
{
  for (int i = 0; i < size; i++) {
    const Node parent = parents[i];
    const uint8_t depth = parent.depth;

    if (depth == N) {
      *exploredSol += 1;
    }
    for (int j = depth; j < N; j++) {
      if (evals[j + i * N] == 1) {
        Node child;
        memcpy(child.board, parent.board, N * sizeof(uint8_t));
        swap(&child.board[depth], &child.board[j]);
        child.depth = depth + 1;
        pushBack(pool, child);
        *exploredTree += 1;
      }
    }
  }
}

// Single-core single-GPU N-Queens search.
void nqueens_search(const int N, const int G, const int minSize, const int maxSize,
  unsigned long long int* exploredTree, unsigned long long int* exploredSol)
{
  Node root;
  initRoot(&root, N);

  SinglePool pool;
  initSinglePool(&pool);

  pushBack(&pool, root);

  int count = 0;

  while (1) {
    int hasWork = 0;
    Node parent = popBack(&pool, &hasWork);
    if (!hasWork) break;

    decompose(N, G, parent, exploredTree, exploredSol, &pool);

    int poolSize = MIN(pool.size, maxSize);

    // If 'poolSize' is sufficiently large, we offload the pool on GPU.
    if (poolSize >= minSize) {
      Node* parents = (Node*)malloc(poolSize * sizeof(Node));
      for (int i = 0; i < poolSize; i++) {
        int hasWork = 0;
        parents[i] = popBack(&pool, &hasWork);
        if (!hasWork) break;
      }

      const int evalsSize = N * poolSize;
      uint8_t* evals = (uint8_t*)malloc(evalsSize * sizeof(uint8_t));

      Node* parents_d;
      uint8_t* evals_d;

      hipMalloc(&parents_d, poolSize * sizeof(Node));
      hipMalloc(&evals_d, evalsSize * sizeof(uint8_t));
      hipMemcpy(parents_d, parents, poolSize * sizeof(Node), hipMemcpyHostToDevice);

      int nbBlocks = ceil((double)evalsSize / BLOCK_SIZE);

      count += 1;
      evaluate_gpu<<<nbBlocks, BLOCK_SIZE>>>(N, G, parents_d, evals_d, evalsSize);
      hipDeviceSynchronize();

      hipMemcpy(evals, evals_d, evalsSize * sizeof(uint8_t), hipMemcpyDeviceToHost);

      hipFree(parents_d);
      hipFree(evals_d);

      generate_children(N, parents, poolSize, evals, exploredTree, exploredSol, &pool);

      free(parents);
      free(evals);
    }
  }

  printf("\nExploration terminated.\n");
  printf("Cuda kernel calls: %d\n", count);

  deleteSinglePool(&pool);
}

int main(int argc, char* argv[])
{
  int N, G, minSize, maxSize;
  parse_parameters(argc, argv, &N, &G, &minSize, &maxSize);
  print_settings(N, G);

  unsigned long long int exploredTree = 0;
  unsigned long long int exploredSol = 0;

  clock_t startTime = clock();

  nqueens_search(N, G, minSize, maxSize, &exploredTree, &exploredSol);

  clock_t endTime = clock();
  double totalTime = (double)(endTime - startTime) / CLOCKS_PER_SEC;

  print_results(exploredTree, exploredSol, totalTime);

  return 0;
}
