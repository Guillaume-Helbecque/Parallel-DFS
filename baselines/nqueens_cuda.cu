#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "hip/hip_runtime.h"

#define BLOCK_SIZE 64

#define MIN(a, b) ((a) < (b) ? (a) : (b))

/*******************************************************************************
Implementation of N-Queens Nodes.
*******************************************************************************/

#define MAX_QUEENS 15

typedef struct
{
  int depth;
  int board[MAX_QUEENS];
} Node;

void initRoot(Node* root, const int N)
{
  root->depth = 0;
  for (int i = 0; i < N; i++) {
    root->board[i] = i;
  }
}

/*******************************************************************************
Implementation of a dynamic-sized single pool data structure.
Its initial capacity is 1024, and we reallocate a new container with double
the capacity when it is full. Since we perform only DFS, it only supports
'pushBack' and 'popBack' operations.
*******************************************************************************/

#define CAPACITY 1024

typedef struct
{
  Node* elements;
  int capacity;
  int size;
} SinglePool;

void initSinglePool(SinglePool* pool)
{
  pool->elements = (Node*)malloc(CAPACITY * sizeof(Node));
  pool->capacity = CAPACITY;
  pool->size = 0;
}

void pushBack(SinglePool* pool, Node node)
{
  if (pool->size >= pool->capacity) {
    pool->capacity *= 2;
    pool->elements = (Node*)realloc(pool->elements, pool->capacity * sizeof(Node));
  }

  pool->elements[pool->size++] = node;
}

Node popBack(SinglePool* pool, int* hasWork)
{
  if (pool->size > 0) {
    *hasWork = 1;
    return pool->elements[--pool->size];
  }

  return (Node){0};
}

void deleteSinglePool(SinglePool* pool)
{
  free(pool->elements);
}

/*******************************************************************************
Implementation of the single-core single-GPU N-Queens search.
*******************************************************************************/

void parse_parameters(int argc, char* argv[], int* N, int* G, int* minSize, int* maxSize)
{
  if (argc != 5) {
    printf("Usage: %s <N> <g> <minSize> <maxSize>\n", argv[0]);
    exit(0);
  }

  *N = atoi(argv[1]);
  *G = atoi(argv[2]);
  *minSize = atoi(argv[3]);
  *maxSize = atoi(argv[4]);

  if ((*N <= 0) || (*G <= 0) || (*minSize <= 0) || (*maxSize <= 0)) {
    printf("All parameters must be positive integers.\n");
    exit(0);
  }
}

void print_settings(const int N, const int G)
{
  printf("\n=================================================\n");
  printf("Resolution of the %d-Queens instance using C+CUDA\n", N);
  printf("  with %d safety check(s) per evaluation\n", G);
  printf("=================================================\n");
}

void print_results(const int exploredTree, const int exploredSol, const double timer)
{
  printf("\n=================================================\n");
  printf("Size of the explored tree: %d\n", exploredTree);
  printf("Number of explored solutions: %d\n", exploredSol);
  printf("Elapsed time: %.4f [s]\n", timer);
  printf("=================================================\n");
}

void swap(int* a, int* b)
{
  int tmp = *b;
  *b = *a;
  *a = tmp;
}

// Check queen's safety.
int isSafe(const int G, const int* board, const int queen_num, const int row_pos)
{
  for (int g = 0; g < G; g++) {
    for (int i = 0; i < queen_num; i++) {
      const int other_row_pos = board[i];

      if (other_row_pos == row_pos - (queen_num - i) ||
          other_row_pos == row_pos + (queen_num - i)) {
        return 0;
      }
    }
  }

  return 1;
}

// Evaluate and generate children nodes on CPU.
void decompose(const int N, const int G, const Node parent, int* tree_loc, int* num_sol, SinglePool* pool)
{
  const int depth = parent.depth;

  if (depth == N) {
    *num_sol += 1;
  }
  for (int j = depth; j < N; j++) {
    if (isSafe(G, parent.board, depth, parent.board[j])) {
      Node child;
      memcpy(child.board, parent.board, N * sizeof(int));
      swap(&child.board[depth], &child.board[j]);
      child.depth = depth + 1;
      pushBack(pool, child);
      *tree_loc += 1;
    }
  }
}

// Evaluate a bulk of parent nodes on GPU.
__global__ void evaluate_gpu(const int N, const int G, const Node* parents_d, int* status_d, const int size)
{
  int threadId = blockIdx.x * blockDim.x + threadIdx.x;

  if (threadId < size) {
    const int parentId = threadId / N;
    const int k = threadId % N;
    const Node parent = parents_d[parentId];
    const int depth = parent.depth;

    status_d[threadId] = 1;

    // If child 'k' is not scheduled, we evaluate its safety 'G' times, otherwise 0.
    const int G_notScheduled = G * (k >= depth);
    for (int g = 0; g < G_notScheduled; g++) {
      for (int i = 0; i < depth; i++) {
        const int other_row_pos = parent.board[i];
        const int isNotSafe = (other_row_pos == parent.board[k] - (depth - i) ||
                               other_row_pos == parent.board[k] + (depth - i));

        status_d[threadId] *= (1 - isNotSafe);
      }
    }
  }
}

// Generate children nodes (evaluated by GPU) on CPU.
void generate_children(const int N, const Node* parents, const int size, const int* evals,
  int* exploredTree, int* exploredSol, SinglePool* pool)
{
  for (int i = 0; i < size; i++) {
    const Node parent = parents[i];
    const int depth = parent.depth;

    if (depth == N) {
      *exploredSol += 1;
    }
    for (int j = depth; j < N; j++) {
      if (evals[j + i * N] == 1) {
        Node child;
        memcpy(child.board, parent.board, N * sizeof(int));
        swap(&child.board[depth], &child.board[j]);
        child.depth = depth + 1;
        pushBack(pool, child);
        *exploredTree += 1;
      }
    }
  }
}

// Single-core single-GPU N-Queens search.
void nqueens_search(const int N, const int G, const int minSize, const int maxSize,
  int* exploredTree, int* exploredSol)
{
  Node root;
  initRoot(&root, N);

  SinglePool pool;
  initSinglePool(&pool);

  pushBack(&pool, root);

  int count = 0;

  while (1) {
    int hasWork = 0;
    Node parent = popBack(&pool, &hasWork);
    if (!hasWork) break;

    decompose(N, G, parent, exploredTree, exploredSol, &pool);

    int poolSize = MIN(pool.size, maxSize);

    // If 'poolSize' is sufficiently large, we offload the pool on GPU.
    if (poolSize >= minSize) {
      Node* parents = (Node*)malloc(poolSize * sizeof(Node));
      for (int i = 0; i < poolSize; i++) {
        int hasWork = 0;
        parents[i] = popBack(&pool, &hasWork);
        if (!hasWork) break;
      }

      const int evalsSize = N * poolSize;
      int* evals = (int*)malloc(evalsSize * sizeof(int));

      Node* parents_d;
      int* status_d;

      hipMalloc(&parents_d, poolSize * sizeof(Node));
      hipMalloc(&status_d, evalsSize * sizeof(int));
      hipMemcpy(parents_d, parents, poolSize * sizeof(Node), hipMemcpyHostToDevice);

      int nbBlocks = ceil((double)evalsSize / BLOCK_SIZE);

      count += 1;
      evaluate_gpu<<<nbBlocks, BLOCK_SIZE>>>(N, G, parents_d, status_d, evalsSize);
      hipDeviceSynchronize();

      hipMemcpy(evals, status_d, evalsSize * sizeof(int), hipMemcpyDeviceToHost);

      hipFree(parents_d);
      hipFree(status_d);

      generate_children(N, parents, poolSize, evals, exploredTree, exploredSol, &pool);

      free(parents);
      free(evals);
    }
  }

  printf("\nExploration terminated.\n");
  printf("Cuda kernel calls: %d\n", count);

  deleteSinglePool(&pool);
}

int main(int argc, char* argv[])
{
  int N, G, minSize, maxSize;
  parse_parameters(argc, argv, &N, &G, &minSize, &maxSize);
  print_settings(N, G);

  int exploredTree = 0;
  int exploredSol = 0;

  clock_t startTime = clock();

  nqueens_search(N, G, minSize, maxSize, &exploredTree, &exploredSol);

  clock_t endTime = clock();
  double totalTime = (double)(endTime - startTime) / CLOCKS_PER_SEC;

  print_results(exploredTree, exploredSol, totalTime);

  return 0;
}
