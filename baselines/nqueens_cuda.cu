#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime.h"

#define MIN(a, b) ((a) < (b) ? (a) : (b))

// Definition of the N-Queens Node type
#define MAX_QUEENS 15

typedef struct
{
  int depth;
  int board[MAX_QUEENS];
} Node;

// Implementation of a basic single pool
#define CAPACITY 1024

typedef struct
{
  Node* elements;
  int capacity;
  int size;
} SinglePool;

void initSinglePool(SinglePool* pool)
{
  pool->elements = (Node*)malloc(CAPACITY * sizeof(Node));
  pool->capacity = CAPACITY;
  pool->size = 0;
}

void pushBack(SinglePool* pool, Node node)
{
  if (pool->size < pool->capacity) {
    pool->elements[pool->size] = node;
    pool->size++;
  } else {
    Node* tmp = (Node*)malloc(pool->capacity * sizeof(Node));
    memcpy(tmp, pool->elements, pool->capacity * sizeof(Node));
    free(pool->elements);
    pool->elements = (Node*)malloc(2 * pool->capacity * sizeof(Node));
    for (int i = 0; i < pool->capacity; i++) {
      pool->elements[i] = tmp[i];
    }
    pool->capacity = 2 * pool->capacity;

    pool->elements[pool->size] = node;
    pool->size++;
  }
}

Node popBack(SinglePool* pool, int* hasWork)
{
  if (pool->size > 0) {
    Node node = pool->elements[pool->size - 1];
    pool->size--;
    *hasWork = 1;
    return node;
  }

  Node node_default;
  return node_default;
}

void clearSinglePool(SinglePool* pool)
{
  free(pool->elements);
  pool->capacity = CAPACITY;
  pool->size = 0;
}

// Implementation of the N-Queens search

void parse_parameters(int argc, char* argv[], int* N, int* G, int* minSize, int* maxSize)
{
  if (argc != 5) {
    printf("Usage: %s <N> <g> <minSize> <maxSize>\n", argv[0]);
    exit(0);
  }

  *N = atoi(argv[1]);
  *G = atoi(argv[2]);
  *minSize = atoi(argv[3]);
  *maxSize = atoi(argv[4]);

  if ((*N <= 0) || (*G <= 0) || (*minSize <= 0) || (*maxSize <= 0)) {
    printf("All parameters must be positive integers.\n");
    exit(0);
  }
}

void print_settings(const int N, const int G)
{
  printf("\n=================================================\n");
  printf("Resolution of the %d-Queens instance using C+CUDA\n", N);
  printf("  with %d safety check(s) per evaluation\n", G);
  printf("=================================================\n");
}

void print_results(const int exploredTree, const int exploredSol, const double timer)
{
  printf("\n=================================================\n");
  printf("Size of the explored tree: %d\n", exploredTree);
  printf("Number of explored solutions: %d\n", exploredSol);
  printf("Elapsed time: %.4f [s]\n", timer);
  printf("=================================================\n");
}

// Swap two integers
void swap(int* a, int* b)
{
  int tmp = *b;
  *b = *a;
  *a = tmp;
}

// Check queen's safety
int isSafe(const int G, const int* board, const int queen_num, const int row_pos)
{
  for (int g = 0; g < G; g++) {
    for (int i = 0; i < queen_num; i++) {
      const int other_row_pos = board[i];

      if (other_row_pos == row_pos - (queen_num - i) ||
          other_row_pos == row_pos + (queen_num - i)) {
        return 0;
      }
    }
  }

  return 1;
}

void decompose(const int N, const int G, const Node parent, int* tree_loc, int* num_sol, SinglePool* pool)
{
  const int depth = parent.depth;

  if (depth == N) {
    *num_sol += 1;
  }
  for (int j = depth; j < N; j++) {
    if (isSafe(G, parent.board, depth, parent.board[j])) {
      Node child;
      for (int i = 0; i < N; i++) {
        child.board[i] = parent.board[i];
      }
      swap(&child.board[depth], &child.board[j]);
      child.depth = depth + 1;
      pushBack(pool, child);
      *tree_loc += 1;
    }
  }
}

// Evaluate a bulk of parent nodes on GPU
__global__ void evaluate_gpu(const int N, const int G, const Node* parents_d, int* status_d, const int size)
{
  int pid = blockIdx.x * blockDim.x + threadIdx.x;

  if (pid < size) {
    const int parentId = pid / N;
    const int k = pid % N;
    const Node parent = parents_d[parentId];
    const int depth = parent.depth;

    status_d[pid] = 1;

    const int notScheduled = (int)(k >= depth);
    for (int g = 0; g < (notScheduled*G - (1-notScheduled)); g++) {
      for (int i = 0; i < depth; i++) {
        const int other_row_pos = parent.board[i];
        const int isNotSafe = (other_row_pos == parent.board[k] - (depth - i) ||
          other_row_pos == parent.board[k] + (depth - i));

        status_d[pid] = isNotSafe * (-1) + (1-isNotSafe) * status_d[pid];
      }
    }
  }
}

void process_children(const int N, const Node* parents, const int size, const int* evals,
  int* exploredTree, int* exploredSol, SinglePool* pool)
{
  for (int i = 0; i < size; i++) {
    const Node parent = parents[i];
    const int depth = parent.depth;

    if (depth == N) {
      *exploredSol += 1;
    }
    for (int j = depth; j < N; j++) {
      if (evals[j + i * N] == 1) {
        Node child;
        for (int i = 0; i < N; i++) {
          child.board[i] = parent.board[i];
        }
        swap(&child.board[depth], &child.board[j]);
        child.depth = depth + 1;
        pushBack(pool, child);
        *exploredTree += 1;
      }
    }
  }
}

void nqueens_search(const int N, const int G, const int minSize, const int maxSize,
  int* exploredTree, int* exploredSol)
{
  Node root;
  root.depth = 0;
  for (int i = 0; i < N; i++) {
    root.board[i] = i;
  }

  SinglePool pool;
  initSinglePool(&pool);
  pushBack(&pool, root);

  while (1) {
    int hasWork = 0;
    Node parent = popBack(&pool, &hasWork);
    if (!hasWork) {
      break;
    }

    decompose(N, G, parent, exploredTree, exploredSol, &pool);

    int poolSize = MIN(pool.size, maxSize);

    if (poolSize >= minSize) {
      Node* parents = (Node*)malloc(poolSize * sizeof(Node));
      for (int i = 0; i < poolSize; i++) {
        int hasWork = 0;
        parents[i] = popBack(&pool, &hasWork);
        if (!hasWork) {
          break;
        }
      }
      int* evals = (int*)malloc(N * poolSize * sizeof(int));

      Node* parents_d;
      int* status_d;

      // Offload node evaluation on GPU
      hipMalloc(&parents_d, poolSize * sizeof(Node));
      hipMalloc(&status_d, N * poolSize * sizeof(int));
      hipMemcpy(parents_d, parents, poolSize * sizeof(Node), hipMemcpyHostToDevice);

      int blockSize = 64;
      int nbBlocks = (N * poolSize / blockSize) + (((N * poolSize) % blockSize) == 0 ? 0 : 1);

      evaluate_gpu<<<nbBlocks, blockSize>>>(N, G, parents_d, status_d, N * poolSize);

      hipMemcpy(evals, status_d, N * poolSize * sizeof(int), hipMemcpyDeviceToHost);

      hipFree(parents_d);
      hipFree(status_d);

      process_children(N, parents, poolSize, evals, exploredTree, exploredSol, &pool);

      free(parents);
      free(evals);
    }
  }
  clearSinglePool(&pool);
}

int main(int argc, char* argv[])
{
  int N, G, minSize, maxSize;
  parse_parameters(argc, argv, &N, &G, &minSize, &maxSize);
  print_settings(N, G);

  int exploredTree = 0;
  int exploredSol = 0;

  clock_t startTime = clock();

  nqueens_search(N, G, minSize, maxSize, &exploredTree, &exploredSol);

  clock_t endTime = clock();
  double totalTime = (double)(endTime - startTime) / CLOCKS_PER_SEC;

  print_results(exploredTree, exploredSol, totalTime);

  return 0;
}
